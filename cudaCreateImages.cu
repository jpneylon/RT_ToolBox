#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <cstring>
#include <hip/hip_runtime_api.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

#include "defs.h"
#include "cudaCreateImages.cuh"



extern "C" void cudaCreateImageSet( FLOAT_GRID *datafloat, IMAGE_SET *datachar, float3 rgb, int3 outSize, int upscaler, uint switcher)
{
    /////////////////// Bind Inputs to 3D Texture Arrays //////////////////////////////////////////////
        hipArray *Vol_Array=0;
		hipExtent VolumeSize	            =	make_hipExtent((float)datafloat->size.x, (float)datafloat->size.y, (float)datafloat->size.z);
		hipChannelFormatDesc channelDesc	=	hipCreateChannelDesc<float>();
		hipMalloc3DArray(&Vol_Array, &channelDesc, VolumeSize);

		hipMemcpy3DParms CopyParams = {0};
		CopyParams.srcPtr	    =	make_hipPitchedPtr((void*)datafloat->matrix,VolumeSize.width*sizeof(float), VolumeSize.width, VolumeSize.height);
		CopyParams.dstArray	    =	Vol_Array;
		CopyParams.extent	    =	VolumeSize;
		CopyParams.kind		    =	hipMemcpyHostToDevice;
		hipMemcpy3D(&CopyParams);

		texImageSrc.normalized		=	false;
		texImageSrc.filterMode		=	hipFilterModeLinear;
		texImageSrc.addressMode[0]	=	hipAddressModeClamp;
		texImageSrc.addressMode[1]	=	hipAddressModeClamp;
		texImageSrc.addressMode[2]	=	hipAddressModeClamp;

		hipBindTextureToArray(texImageSrc, Vol_Array, channelDesc);
    //////////////////////////////////////////////////////////////////////////////////////////////

    int imageSize = 3*outSize.x*outSize.y*sizeof( unsigned char );
    uchar *sliceData;
    checkCudaErrors( hipMalloc( (void**)&sliceData, imageSize ));

    dim3 block(16,16);
    int2 gridSize; gridSize.x = outSize.x/16; gridSize.y = outSize.y/16;
    if ( outSize.x % 16 > 0 ) gridSize.x++;
    if ( outSize.y % 16 > 0 ) gridSize.y++;
    dim3 grid(gridSize.x,gridSize.y);

    for (int k=0; k<outSize.z; k++)
    {
        datachar->anatomy[k].pixels = new unsigned char[3*outSize.x*outSize.y];
        checkCudaErrors(hipMemset(sliceData,0,imageSize));

        if (switcher == 0)
            createAxialImage_kernel<<<grid,block>>>( sliceData,
                                                     datafloat->min,
                                                     datafloat->max,
                                                     datafloat->size,
                                                     datafloat->voxel,
                                                     rgb,
                                                     upscaler, k,
                                                     outSize );
        else if (switcher == 1)
            createSagittalImage_kernel<<<grid,block>>>( sliceData,
                                                        datafloat->min,
                                                        datafloat->max,
                                                        datafloat->size,
                                                        datafloat->voxel,
                                                        rgb,
                                                        upscaler, k,
                                                        outSize );
        else if (switcher == 2)
            createCoronalImage_kernel<<<grid,block>>>(  sliceData,
                                                        datafloat->min,
                                                        datafloat->max,
                                                        datafloat->size,
                                                        datafloat->voxel,
                                                        rgb,
                                                        upscaler, k,
                                                        outSize );
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipMemcpy(datachar->anatomy[k].pixels,sliceData,imageSize,hipMemcpyDeviceToHost));
        //printf("\n Slice %d created...",k);
    }
    checkCudaErrors(hipFree(sliceData));
    checkCudaErrors(hipUnbindTexture(texImageSrc));
    checkCudaErrors(hipFreeArray(Vol_Array));
}


extern "C" void cudaCreateOrientationImage( FLOAT_GRID *datafloat, IMAGE_SET *datachar, float3 rgb, int3 outSize, int upscaler)
{
    /////////////////// Bind Inputs to 3D Texture Arrays //////////////////////////////////////////////
        printf(" Using texture memory...");
        hipArray *Vol_Array=0;
		hipExtent VolumeSize	            =	make_hipExtent((float)datafloat->size.x, (float)datafloat->size.y, (float)datafloat->size.z);
		hipChannelFormatDesc channelDesc	=	hipCreateChannelDesc<float>();
		hipMalloc3DArray(&Vol_Array, &channelDesc, VolumeSize);

		hipMemcpy3DParms CopyParams = {0};
		CopyParams.srcPtr	    =	make_hipPitchedPtr((void*)datafloat->matrix,VolumeSize.width*sizeof(float), VolumeSize.width, VolumeSize.height);
		CopyParams.dstArray	    =	Vol_Array;
		CopyParams.extent	    =	VolumeSize;
		CopyParams.kind		    =	hipMemcpyHostToDevice;
		hipMemcpy3D(&CopyParams);

		texImageSrc.normalized		=	false;
		texImageSrc.filterMode		=	hipFilterModeLinear;
		texImageSrc.addressMode[0]	=	hipAddressModeClamp;
		texImageSrc.addressMode[1]	=	hipAddressModeClamp;
		texImageSrc.addressMode[2]	=	hipAddressModeClamp;

		hipBindTextureToArray(texImageSrc, Vol_Array, channelDesc);
    //////////////////////////////////////////////////////////////////////////////////////////////

    int imageSize = 3*outSize.x*outSize.y*sizeof( uchar );
    uchar *sliceData;
    checkCudaErrors( hipMalloc( (void**)&sliceData, imageSize ));
    checkCudaErrors( hipMemset( sliceData, 0, imageSize ) );

    dim3 block(16,16);
    int2 gridSize; gridSize.x = outSize.x/16; gridSize.y = outSize.y/16;
    if ( outSize.x % 16 > 0 ) gridSize.x++;
    if ( outSize.y % 16 > 0 ) gridSize.y++;
    dim3 grid(gridSize.x,gridSize.y);

    createOrientationImage_kernel<<<grid,block>>>(  sliceData,
                                                    datafloat->min,
                                                    datafloat->max,
                                                    datafloat->size,
                                                    datafloat->voxel,
                                                    rgb,
                                                    upscaler, datafloat->size.x/2,
                                                    outSize );
    hipDeviceSynchronize();
    getLastCudaError("Kernel execution failed");

    datachar->anatomy[0].pixels = new unsigned char[3*outSize.x*outSize.y];
    checkCudaErrors(hipMemcpy(datachar->anatomy[0].pixels,sliceData,imageSize,hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(sliceData));
    checkCudaErrors(hipUnbindTexture(texImageSrc));
    checkCudaErrors(hipFreeArray(Vol_Array));
}


extern "C" void cudaCreateOverlayImageSet( FLOAT_GRID *overlay, IMAGE_SET *datachar, float RX_DOSE, float alpha, int upscaler, int switcher)
{
    /////////////////// Bind Inputs to 3D Texture Arrays //////////////////////////////////////////////
        printf("\n Using texture memory...\n"); fflush(stdout);
        hipArray *Over_Array;
		hipExtent VolumeSize	            =	make_hipExtent((float)overlay->size.x, (float)overlay->size.y, (float)overlay->size.z);
		hipChannelFormatDesc channelDesc	=	hipCreateChannelDesc<float>();
		hipMalloc3DArray(&Over_Array, &channelDesc, VolumeSize);

		hipMemcpy3DParms CopyParamsOver = {0};
		CopyParamsOver.srcPtr	    =	make_hipPitchedPtr((void*)overlay->matrix,VolumeSize.width*sizeof(float), VolumeSize.width, VolumeSize.height);
		CopyParamsOver.dstArray	    =	Over_Array;
		CopyParamsOver.extent	    =	VolumeSize;
		CopyParamsOver.kind		    =	hipMemcpyHostToDevice;
		hipMemcpy3D(&CopyParamsOver);

		texOverlaySrc.normalized		=	false;
		texOverlaySrc.filterMode		=	hipFilterModeLinear;
		texOverlaySrc.addressMode[0]	=	hipAddressModeClamp;
		texOverlaySrc.addressMode[1]	=	hipAddressModeClamp;
		texOverlaySrc.addressMode[2]	=	hipAddressModeClamp;

		hipBindTextureToArray(texOverlaySrc, Over_Array, channelDesc);
    //////////////////////////////////////////////////////////////////////////////////////////////

    int X = upscaler*overlay->size.x;
    int Y = upscaler*overlay->size.y;
    int Z = overlay->size.z;

    int imageSize = 3*X*Y*sizeof( uchar );
    printf("\n IMAGE SIZE: 3 x %d x %d x %lu = %d", X, Y, sizeof(uchar), imageSize);

    uchar *sliceData;
    checkCudaErrors( hipMalloc( (void**)&sliceData, imageSize ));

    dim3 block(16,16);
    int2 gridSize; gridSize.x = X/16; gridSize.y = Y/16;
    if ( X%16 > 0 ) gridSize.x++;
    if ( Y%16 > 0 ) gridSize.y++;
    dim3 grid(gridSize.x,gridSize.y);

    for (int k=0; k<Z; k++)
    {
        checkCudaErrors(hipMemset(sliceData,0,imageSize));

        if (switcher == 0){
            createDoseImage_kernel<<<grid,block>>>( sliceData,
                                                    RX_DOSE,
                                                    alpha,
                                                    overlay->size,
                                                    overlay->voxel,
                                                    upscaler, k );
        }
        else if (switcher == 1){
            createGammaImage_kernel<<<grid,block>>>(sliceData,
                                                    alpha,
                                                    overlay->size,
                                                    overlay->voxel,
                                                    upscaler, k );
        }
        else if (switcher == 2){
            createJacobImage_kernel<<<grid,block>>>(sliceData,
                                                    alpha,
                                                    overlay->size,
                                                    overlay->voxel,
                                                    upscaler, k );
        }
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");

        datachar->overlay[k].pixels = new unsigned char[3*X*Y];
        checkCudaErrors(hipMemcpy(datachar->overlay[k].pixels,sliceData,imageSize,hipMemcpyDeviceToHost));
    }

    checkCudaErrors(hipFree(sliceData));
    checkCudaErrors(hipUnbindTexture(texOverlaySrc));
    checkCudaErrors(hipFreeArray(Over_Array));
}





